#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <locale.h>
#include <hip/hip_runtime.h>

void getmaxcu(unsigned int * number, unsigned int * maximum, int size);

__global__
void getmaxcu(unsigned int * number, unsigned int * maximum, int size) {
    extern __shared__ unsigned int data[];

    unsigned int t_id = threadIdx.x;
    unsigned int index = blockIdx.x * (blockDim.x) + threadIdx.x;

    data[t_id] = num[index];

    if (index >= size) {
        data[t_id] = 0;
    }

    __syncthreads();

    int loop_index;
    for (loop_index = blockDim.x / 2; loop_index > 0; loop_index >>= 1) {

        __syncthreads();

        if (t_id < loop_index) {
            if (data[t_id] < data[t_id + loop_index]) {
                data[t_id] = data[t_id + loop_index];
            }
        }
    }

    if (t_id == 0) {
        maximum[blockIdx.x] = data[0];
    }
}
