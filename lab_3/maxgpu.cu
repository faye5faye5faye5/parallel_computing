#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <locale.h>
#include <hip/hip_runtime.h>

void getmaxcu(unsigned int * number, unsigned int * maximum, int size);

int main(int argc, char * argv[]) {
    unsigned int size;
    unsigned int loop_index;
    unsigned int * number_arr;

    if (argc != 2) {
        printf("Not enough command line arguments. Try again.\n");
        exit(1);
    }

    size = atoi(argv[1]);

    number_arr = (unsigned int *) malloc(sizeof(unsigned int) * size);

    if (!number_arr) {
        printf("Something went wrong when allocating space for number_arr. Try again.\n");
        exit(1);
    }

    srand(time(NULL));

    for (loop_index = 0; loop_index < size; loop_index += 1) {
        number_arr[loop_index] = rand() % size;
    }

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);

    int threads_per_block = device_properties.maxThreadsPerBlock;
    int num_threads;

    if (threads_per_block > size) {
        num_threads = size;
    }
    else {
        num_threads = threads_per_block;
    }

    int mod_size = num_threads;

    if (size % threads_per_block != 0) {
        mod_size = (size / threads_per_block + 1) * threads_per_block;
    }
    else {
        mod_size = size;
    }

    unsigned int * mod_number_arr = (unsigned int *) malloc(sizeof(unsigned int) * mod_size);

    for (loop_index = 0; loop_index < mod_size; loop_index += 1) {
        if (loop_index < size) {
            mod_number_arr[loop_index] = number_arr[loop_index];
        }
        else {
            mod_number_arr[loop_index] = 0;
        }
    }

    int num_blocks = mod_size / threads_per_block;
    unsigned int * device_arr;

    hipMalloc((void **) &device_arr, sizeof(unsigned int) * mod_size);
    hipMemcpy(device_arr, mod_number_arr, sizeof(unsigned int) * mod_size, hipMemcpyHostToDevice);

    unsigned int * device_max;

    hipMalloc((void **) &device_max, sizeof(unsigned int) * num_blocks);

    unsigned int * host_max = (unsigned int *) malloc(sizeof(unsigned int) * num_blocks);

    do {
        num_blocks = ceil((float) mod_size / (float) threads_per_block);
        getmaxcu<<num_blocks, threads_per_block, sizeof(unsigned int) * threads_per_block>>(device_arr, device_max, mod_size);
        mod_size = num_blocks;
        device_arr = device_max;
    } while (num_blocks > 1);

    hipMemcpy(host_max, device_max, sizeof(unsigned int) * num_blocks, hipMemcpyDeviceToHost);
    printf("The maximum number is %u\n", host_max[0]);

    hipFree(device_arr);
    hipFree(device_max);
    free(number_arr);
    exit(0);
}

__global__
void getmaxcu(unsigned int * number, unsigned int * maximum, int size) {
    extern __shared__ unsigned int data[];

    unsigned int t_id = threadIdx.x;
    unsigned int index = blockIdx.x * (blockDim.x) + threadIdx.x;

    data[t_id] = num[index];

    if (index >= size) {
        data[t_id] = 0;
    }

    __syncthreads();

    int loop_index;
    for (loop_index = blockDim.x / 2; loop_index > 0; loop_index >>= 1) {

        __syncthreads();

        if (t_id < loop_index) {
            if (data[t_id] < data[t_id + loop_index]) {
                data[t_id] = data[t_id + loop_index];
            }
        }
    }

    if (t_id == 0) {
        maximum[blockIdx.x] = data[0];
    }
}
