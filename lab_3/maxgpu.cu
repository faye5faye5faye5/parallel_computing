#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <locale.h>
#include <hip/hip_runtime.h>

__global__
void getmaxcu(unsigned int * number, unsigned int * maximum, int size) {
    extern __shared__ unsigned int data[];

    unsigned int t_id = threadIdx.x;
    unsigned int index = blockIdx.x * (blockDim.x) + threadIdx.x;

    data[t_id] = num[index];

    if (index >= size) {
        data[t_id] = 0;
    }
}
